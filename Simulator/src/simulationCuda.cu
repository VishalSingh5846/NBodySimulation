#include "hip/hip_runtime.h"
#include <assert.h>
#include <iostream>
#include <string.h>
#include <omp.h>
#include<math.h>
#include <time.h>
#include<chrono>
using namespace std;

#define BENCHMARK true
#define TIME_STEP 0.0005

#define DEVICE_MEM true

#define MASS_MAX 20
#define MASS_MIN 10
#define RADIUS_MAX 0.01
#define RADIUS_MIN 0.01
#define X_MAX 2
#define X_MIN -2
#define Y_MAX 2
#define Y_MIN -2
#define Z_MAX -5
#define Z_MIN -5
#define VEL_MAX 0
#define VEL_MIN 0
#define ACC_MAX 0
#define ACC_MIN 0
#define CUDA_CALL(x) do {						\
    hipError_t ____rc = (x);					\
    assert(____rc == hipSuccess);					\
  } while (0)

int ITERATIONS;
int N_BODY;

struct Vec3{
    double x;
    double y;
    double z;
};
__device__ void magnitude(Vec3* vec, double* mag){
    *mag = sqrt(vec->x*vec->x + vec->y*vec->y + vec->z*vec->z);
}
__device__ void normalize(Vec3* vec){
    double mag;
    magnitude(vec, &mag);
    vec->x /= mag;
    vec->y /= mag;
    vec->z /= mag;
}
__device__ void multiply(Vec3* vec, double k){
    vec->x *= k;
    vec->y *= k;
    vec->z *= k;
}
__device__ void add(Vec3* vec1, Vec3* vec2){
    vec1->x += vec2->x;
    vec1->y += vec2->y;
    vec1->z += vec2->z;
}
void print(const char* str, Vec3* vec){
    printf("%s: (%f,%f,%f)\n",str,vec->x,vec->y,vec->z);
}

struct Body{
    double mass;
    double radius;
    Vec3 color;
    Vec3 position;
    Vec3 velocity;
    Vec3 acceleration;
    Body() {};
    Body(double mass, double radius, Vec3 color, Vec3 position, Vec3 velocity, Vec3 acceleration): color(color), velocity(velocity), position(position), acceleration(acceleration)  {
        this->mass = mass;
        this->radius = radius;
    };
    void print(const char* id){
        printf("%s -> Mass: %f, radius:%f, Color:(%f,%f,%f), Pos:(%f,%f,%f), Vel:(%f,%f,%f)\n", id,mass, radius, color.x, color.y, color.z, position.x, position.y, position.z, velocity.x, velocity.y, velocity.z);
    }
    void printPos(const char* id){
        printf("%s -> Pos:(%f,%f,%f), Vel:(%f,%f,%f)\n", id, position.x, position.y, position.z, velocity.x, velocity.y, velocity.z);
    }
};

int numThreads;
Body* bodyListH;
Body* bodyList;

__device__ double TimeSpecToSeconds(struct timespec* ts){
    return (double)ts->tv_sec + (double)ts->tv_nsec / 1000000000.0;
}

double generateRandomNumber(double mi,double ma){
    return  mi + (rand() / (1.0 * RAND_MAX)) * (ma - mi);
}

__host__ void generateBodies(Body* bodyList){
    for(unsigned int i=0; i<N_BODY; i++){
        Vec3 posVec;
        Vec3 velVec;
        Vec3 accVec;
        Vec3 colVec;

        colVec.x = rand() * 1.0 / RAND_MAX;
        colVec.y = rand() * 1.0 / RAND_MAX;
        colVec.z = rand() * 1.0 / RAND_MAX;

        posVec.x = generateRandomNumber(X_MIN, X_MAX);
        posVec.y = generateRandomNumber(Y_MIN, Y_MAX);
        posVec.z = generateRandomNumber(Z_MIN, Z_MAX);

        velVec.x = generateRandomNumber(VEL_MIN, VEL_MAX);
        velVec.y = generateRandomNumber(VEL_MIN, VEL_MAX);
        velVec.z = 0;

        accVec.x = generateRandomNumber(ACC_MIN, ACC_MAX);
        accVec.y = generateRandomNumber(ACC_MIN, ACC_MAX);
        accVec.z = 0;


        bodyList[i] = Body(
            generateRandomNumber(MASS_MIN, MASS_MAX),
            generateRandomNumber(RADIUS_MIN, RADIUS_MAX) , 
            colVec, 
            posVec, 
            velVec,  
            accVec
        );
        // bodyList[i].print("Generating Body");
    }
    
    // Body b2(100,0.3,Vec3(0,0,1),Vec3(2,0,-10),Vec3(0,0,0), Vec3(0,0,0));
    // bodyList[0] = b1;
    // bodyList[1] = b2;   
    // bodyList[0].print("Here");
}

__device__ double distance(Vec3 &p1, Vec3 &p2){
    return sqrt((p1.x-p2.x)*(p1.x-p2.x) + (p1.y-p2.y)*(p1.y-p2.y) + (p1.z-p2.z)*(p1.z-p2.z));
}

__device__ void computeForce(Body &b1, Body &b2, Vec3* force){
    double K = 1;
    double dist = distance(b1.position,b2.position);
    double mag = K * b1.mass * b2.mass / (dist * dist + 0.000001);
    force->x = b2.position.x-b1.position.x;
    force->y = b2.position.y-b1.position.y;
    force->z = b2.position.z-b1.position.z;    
    normalize(force);
    // mag = 0;
    multiply(force,mag);
}

__device__ void modifyBodyPositionAndVelocity(Body &b1){
    b1.position.x += ( b1.velocity.x + 0.5 * b1.acceleration.x * TIME_STEP ) * TIME_STEP;
    b1.position.y += ( b1.velocity.y + 0.5 * b1.acceleration.y * TIME_STEP ) * TIME_STEP;
    b1.position.z += ( b1.velocity.z + 0.5 * b1.acceleration.z * TIME_STEP ) * TIME_STEP;

    b1.velocity.x += b1.acceleration.x * TIME_STEP;
    b1.velocity.y += b1.acceleration.y * TIME_STEP;
    b1.velocity.z += b1.acceleration.z * TIME_STEP;
}

__device__ void computeAcceleration(Body &b1, Vec3 force){
    b1.acceleration.x = force.x / b1.mass;
    // printf("Force:%f, Mass:%d, Force")
    b1.acceleration.y = force.y / b1.mass;
    b1.acceleration.z = force.z / b1.mass;
}

__global__ void computeInteractionShared(Body* bodyListGM, int N_BODY, int ITERATIONS){

    // printf(" Thread %d %d %d %d \n",threadIdx.x,blockIdx.x,blockDim.x,gridDim.x);
    int bodyId = threadIdx.x;
    int jump = blockDim.x;
    extern __shared__ Body bodyListS[];

    // for(int i= bodyId; i<N_BODY; i += jump ){
    //     bodyListS[i] = bodyListGM[i];
    // }
    memcpy(bodyListS, bodyListGM, sizeof(Body) * N_BODY);
    
    // __syncthreads();
    // return;

    for(int iter = 0 ; iter<ITERATIONS; iter++){
        
        for(int i = bodyId; i<N_BODY; i += jump ){
            Vec3 netForce;
            netForce.x = 0;
            netForce.y = 0;
            netForce.z = 0;
            Vec3 force;
            for(int j=0; j<N_BODY; j++){
                if(i==j) continue;
                computeForce(bodyListS[i],bodyListS[j], &force);
                add(&netForce, &force);
            }
            // bodyList[i].acceleration.print("ACC BEFORE");
            // netForce.print("Net Force");
            computeAcceleration(bodyListS[i], netForce);
            // bodyList[i].acceleration.print("ACC AFTER");

        }
        
        // __syncthreads();

        // for(int i= bodyId; i<N_BODY; i += jump ){
        //     modifyBodyPositionAndVelocity(bodyListS[i]);
        // }

    }
    for(int i= bodyId; i<N_BODY; i += jump ){
        bodyListGM[i] = bodyListS[i];
    }

}

__global__ void computeInteraction(Body* bodyList, int N_BODY, int ITERATIONS){

    // printf(" Thread %d %d %d %d \n",threadIdx.x,blockIdx.x,blockDim.x,gridDim.x);
    int bodyId = blockDim.x * blockIdx.x + threadIdx.x;
    int jump = blockDim.x * gridDim.x;

    for(int iter = 0 ; iter<ITERATIONS; iter++){
        
        for(int i = bodyId; i<N_BODY; i += jump ){
            Vec3 netForce;
            netForce.x = 0;
            netForce.y = 0;
            netForce.z = 0;
            Vec3 force;
            for(int j=0; j<N_BODY; j++){
                if(i==j) continue;
                computeForce(bodyList[i],bodyList[j], &force);
                add(&netForce, &force);
            }
            // bodyList[i].acceleration.print("ACC BEFORE");
            // netForce.print("Net Force");
            computeAcceleration(bodyList[i], netForce);
            // bodyList[i].acceleration.print("ACC AFTER");

        }
        
        __syncthreads();

        for(int i= bodyId; i<N_BODY; i += jump ){
            modifyBodyPositionAndVelocity(bodyList[i]);
        }
    }
}

void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %u\n", (unsigned int) devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n", (unsigned int) devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %u\n", (unsigned int)  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
    printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %u\n", (unsigned int) devProp.totalConstMem);
    printf("Texture alignment:             %u\n", (unsigned int) devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

void setupCUDA(){
    CUDA_CALL(hipSetDevice(0));

    int devCount;

    
    CUDA_CALL(hipGetDeviceCount(&devCount));
    // CUDA_CALL(hipMalloc(&bodyList, sizeof(Body) * N_BODY));
    
    if(DEVICE_MEM){
        bodyListH = new Body[N_BODY];
        generateBodies(bodyListH);
        CUDA_CALL(hipMalloc(&bodyList, N_BODY * sizeof(Body)));
        CUDA_CALL(hipMemcpy(bodyList, bodyListH, sizeof(Body) * N_BODY, hipMemcpyHostToDevice));

    } else {
        CUDA_CALL(hipMallocManaged(&bodyList, N_BODY * sizeof(Body)));
        generateBodies(bodyList);

    }
    

    
    // generateBodies(bodyListH);



    // Iterate through devices
    // for (int i = 0; i < devCount; ++i)
    // {
    //     // Get device properties
    //     printf("\nCUDA Device #%d\n", i);
    //     hipDeviceProp_t devProp;
    //     hipGetDeviceProperties(&devProp, i);
    //     printDevProp(devProp);
    // }

}

void printBodies(Body* lst){
    for(int i=0;i<N_BODY;i++){
        lst[i].print("");
    }
}


 
int main(int argc, char **argv){
    N_BODY = atoi(argv[1]);
    ITERATIONS = atoi(argv[2]);
    numThreads = atoi(argv[3]);

    int numThreadPerBlock = 128;
    int numBlocks = (numThreads % numThreadPerBlock == 0 ? 0 : 1) +  numThreads / numThreadPerBlock;
    numThreads = numThreads > numThreadPerBlock ? numThreadPerBlock : numThreads;
    printf("%d x %d = %d\n", numBlocks, numThreads, numBlocks*numThreads); 
    setupCUDA();
    // printBodies(bodyListH);
    printf("Total Memory: %.3f MB\n",sizeof(Body) * N_BODY / (1024.0 * 1024.0));
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    // for(int i=0;i<ITERATIONS;i++){
        // printf("-----\nIteration: %d\n",i);
        // for(int j=0;j<N_BODY;j++){
        //     bodyList[j].printPos("");
        // }
        computeInteraction<<<numBlocks,numThreads>>>(bodyList, N_BODY, ITERATIONS);
        // computeInteractionShared<<<1,atoi(argv[3]), sizeof(Body) * N_BODY>>>(bodyList, N_BODY, ITERATIONS);
        if(!DEVICE_MEM) hipDeviceSynchronize();
        if(DEVICE_MEM) CUDA_CALL(hipMemcpy(bodyListH, bodyList, sizeof(Body) * N_BODY, hipMemcpyDeviceToHost));
        // printBodies(bodyListH);
        
    // }

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    
    double elapsed =  std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() / 1000000.0;
    printf("Iteration: %d\nBodies: %d\nTime: %.5f secs\n", ITERATIONS,N_BODY, elapsed);
    return 0;

}

